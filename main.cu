#pragma once

#include "TreeCC.cuh"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
//#define STB_IMAGE_RESIZE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"
//#include "stb_image_resize2.h"


struct Cubo {
	Texture tex;
	Triangulo tris[12];//12
	Triangulo* dtri;
	XYZ v[8] = { XYZ(1,2,1),XYZ(-1,2,1),XYZ(1,2,-1),XYZ(-1,2,-1),XYZ(1,3,1),XYZ(-1,3,1),XYZ(1,3,-1),XYZ(-1,3,-1) };
	Cubo() {
		this->tex.height = 10;
		this->tex.width = 10;
		this->tex.area = 10 * 10;
		Material materials[100];
		//this->tex.materials = new Material[100];
		Material base;
		base.cor = RGBAFLT(0.8, 0.8, 1.0, 1.0);
		base.reflect = 0;
		base.ipermeability = 0;
		base.glow = true;
		for (int i = 0; i < 100; i += 1) {
			base.cor = RGBAFLT(1.0,0.0,0.5,1.0);
			materials[i] = base;
		}
		hipMalloc((void**)&this->tex.materials, 100 * sizeof(Material));
		hipMemcpy(this->tex.materials, materials, 100 * sizeof(Material), hipMemcpyHostToDevice);
		Texture* dt;
		hipMalloc((void**)&dt, sizeof(Texture));
		hipMemcpy(dt, &this->tex,sizeof(Texture), hipMemcpyHostToDevice);
		//mexer vetores
		
		//frente
		this->tris[0].ponto[0] = &this->v[0];
		this->tris[0].ponto[1] = &this->v[1];
		this->tris[0].ponto[2] = &this->v[2];
		
		this->tris[0].vetor[0] = this->v[1] - this->v[0];
		this->tris[0].vetor[1] = this->v[2] - this->v[1];
		this->tris[0].vetor[2] = this->v[0] - this->v[2];
		
		this->tris[1].ponto[0] = &this->v[3];
		this->tris[1].ponto[1] = &this->v[2];
		this->tris[1].ponto[2] = &this->v[1];

		this->tris[1].vetor[0] = this->v[2] - this->v[1];
		this->tris[1].vetor[1] = this->v[3] - this->v[2];
		this->tris[1].vetor[2] = this->v[1] - this->v[2];

		//esquerda
		this->tris[2].ponto[0] = &this->v[1];
		this->tris[2].ponto[1] = &this->v[3];
		this->tris[2].ponto[2] = &this->v[5];

		this->tris[2].vetor[0] = this->v[5] - this->v[1];
		this->tris[2].vetor[1] = this->v[7] - this->v[5];
		this->tris[2].vetor[2] = this->v[1] - this->v[7];

		this->tris[3].ponto[0] = &this->v[7];
		this->tris[3].ponto[1] = &this->v[5];
		this->tris[3].ponto[2] = &this->v[3];

		this->tris[3].vetor[0] = this->v[3] - this->v[1];
		this->tris[3].vetor[1] = this->v[7] - this->v[3];
		this->tris[3].vetor[2] = this->v[1] - this->v[7];

		//direita

		this->tris[4].ponto[0] = &this->v[0];
		this->tris[4].ponto[1] = &this->v[2];
		this->tris[4].ponto[2] = &this->v[4];

		this->tris[4].vetor[0] = this->v[2] - this->v[0];
		this->tris[4].vetor[1] = this->v[6] - this->v[2];
		this->tris[4].vetor[2] = this->v[0] - this->v[6];

		this->tris[5].ponto[0] = &this->v[6];
		this->tris[5].ponto[1] = &this->v[4];
		this->tris[5].ponto[2] = &this->v[2];

		this->tris[5].vetor[0] = this->v[4] - this->v[0];
		this->tris[5].vetor[1] = this->v[6] - this->v[4];
		this->tris[5].vetor[2] = this->v[0] - this->v[6];

		// tras

		this->tris[6].ponto[0] = &this->v[4];
		this->tris[6].ponto[1] = &this->v[5];
		this->tris[6].ponto[2] = &this->v[6];

		this->tris[6].vetor[0] = this->v[5] - this->v[4];
		this->tris[6].vetor[1] = this->v[6] - this->v[5];
		this->tris[6].vetor[2] = this->v[4] - this->v[6];

		this->tris[7].ponto[0] = &this->v[7];
		this->tris[7].ponto[1] = &this->v[6];
		this->tris[7].ponto[2] = &this->v[5];

		this->tris[7].vetor[0] = this->v[6] - this->v[5];
		this->tris[7].vetor[1] = this->v[7] - this->v[6];
		this->tris[7].vetor[2] = this->v[5] - this->v[7];

		//cima

		this->tris[8].ponto[0] = &this->v[0];
		this->tris[8].ponto[1] = &this->v[1];
		this->tris[8].ponto[2] = &this->v[4];

		this->tris[8].vetor[0] = this->v[1] - this->v[0];
		this->tris[8].vetor[1] = this->v[5] - this->v[1];
		this->tris[8].vetor[2] = this->v[0] - this->v[5];

		this->tris[9].ponto[0] = &this->v[5];
		this->tris[9].ponto[1] = &this->v[4];
		this->tris[9].ponto[2] = &this->v[1];

		this->tris[9].vetor[0] = this->v[4] - this->v[0];
		this->tris[9].vetor[1] = this->v[5] - this->v[4];
		this->tris[9].vetor[2] = this->v[0] - this->v[5];

		//baixo

		this->tris[10].ponto[0] = &this->v[2];
		this->tris[10].ponto[1] = &this->v[3];
		this->tris[10].ponto[2] = &this->v[6];

		this->tris[10].vetor[0] = this->v[6] - this->v[2];
		this->tris[10].vetor[1] = this->v[7] - this->v[6];
		this->tris[10].vetor[2] = this->v[2] - this->v[7];

		this->tris[11].ponto[0] = &this->v[7];
		this->tris[11].ponto[1] = &this->v[6];
		this->tris[11].ponto[2] = &this->v[3];

		this->tris[11].vetor[0] = this->v[3] - this->v[2];
		this->tris[11].vetor[1] = this->v[7] - this->v[3];
		this->tris[11].vetor[2] = this->v[2] - this->v[7];

		for (int t = 0; t < 12; t += 1) {
			this->tris[t].tex = dt;
			this->tris[t].w_h.w[0] = 0.5;
			this->tris[t].w_h.w[1] = 0.5;
			this->tris[t].w_h.w[2] = 0.5;
			this->tris[t].w_h.h[0] = 0.5;
			this->tris[t].w_h.h[1] = 0.5;
			this->tris[t].w_h.h[2] = 0.5;
			first::equacao_plano_host(&this->tris[t]);
		}
		XYZ* dv;
		hipMalloc(&dv, 8 * sizeof(XYZ));
		hipMemcpy(dv, this->v, 8 * sizeof(XYZ), hipMemcpyHostToDevice);
		//frente
		this->tris[0].ponto[0] = &dv[0];
		this->tris[0].ponto[1] = &dv[1];
		this->tris[0].ponto[2] = &dv[2];

		

		this->tris[1].ponto[0] = &dv[3];
		this->tris[1].ponto[1] = &dv[2];
		this->tris[1].ponto[2] = &dv[1];

		

		//esquerda
		this->tris[2].ponto[0] = &dv[0];
		this->tris[2].ponto[1] = &dv[2];
		this->tris[2].ponto[2] = &dv[4];

		

		this->tris[3].ponto[0] = &dv[6];
		this->tris[3].ponto[1] = &dv[4];
		this->tris[3].ponto[2] = &dv[2];

		

		//direita

		this->tris[4].ponto[0] = &dv[1];
		this->tris[4].ponto[1] = &dv[3];
		this->tris[4].ponto[2] = &dv[5];

		

		this->tris[5].ponto[0] = &dv[7];
		this->tris[5].ponto[1] = &dv[5];
		this->tris[5].ponto[2] = &dv[3];

		

		// tras

		this->tris[6].ponto[0] = &dv[4];
		this->tris[6].ponto[1] = &dv[5];
		this->tris[6].ponto[2] = &dv[6];

		

		this->tris[7].ponto[0] = &dv[7];
		this->tris[7].ponto[1] = &dv[6];
		this->tris[7].ponto[2] = &dv[5];

		

		//cima

		this->tris[8].ponto[0] = &dv[2];
		this->tris[8].ponto[1] = &dv[3];
		this->tris[8].ponto[2] = &dv[6];

		

		this->tris[9].ponto[0] = &dv[7];
		this->tris[9].ponto[1] = &dv[6];
		this->tris[9].ponto[2] = &dv[3];

		

		//baixo

		this->tris[10].ponto[0] = &dv[0];
		this->tris[10].ponto[1] = &dv[1];
		this->tris[10].ponto[2] = &dv[4];

		

		this->tris[11].ponto[0] = &dv[5];
		this->tris[11].ponto[1] = &dv[4];
		this->tris[11].ponto[2] = &dv[1];

		hipMalloc((void**)&this->dtri, 12 * sizeof(Triangulo));
		hipMemcpy(this->dtri, this->tris, 12 * sizeof(Triangulo), hipMemcpyHostToDevice);
		
	}

};

void Save_to_file(Creation* c) {
	Frame f;
	World w;
	Cam cm;
	
	hipMemcpy(&w, c->world, sizeof(World), hipMemcpyDeviceToHost);
	hipMemcpy(&cm, w.cam, sizeof(Cam), hipMemcpyDeviceToHost);
	hipMemcpy(&f,cm.frame , sizeof(Frame), hipMemcpyDeviceToHost);
	//hipMemcpy(&f, c->world->cam->frame, sizeof(Frame), hipMemcpyDeviceToHost);
	COR* buffer = new COR[4*f.w * f.h];
	hipMemcpy(buffer, f.buffer, 4*f.w * f.h * sizeof(COR), hipMemcpyDeviceToHost);
	//std::cout << f.w<<std::endl;
	stbi_write_png("teste.png", f.w, f.h, 4, buffer, f.w * 4);
}

void Teste(Creation* creation) {
	//Cubo cubo = Cubo();
	//UPXL n_t = 12;
	//World w;// = creation->world;
	//UPXL* n;
	//World* w = creation->world;
	//hipMemcpy(&w->n_tri,&n_t, sizeof(UPXL), hipMemcpyHostToDevice);
	//hipMemcpy(&w, creation->world, sizeof(World*), hipMemcpyDeviceToHost);
	//hipMemcpy(&creation->world->n_tri, &n_t, sizeof(UPXL),  hipMemcpyHostToDevice);
	//hipMemcpy(&w->t_triangulos, cubo.dtri,sizeof(Triangulo*), hipMemcpyDeviceToDevice);
	Render(creation);
	Save_to_file(creation);
	

}





int main() {
	hipSetDevice(0);
	Cubo cubo = Cubo();
	Creation creation(50, 100, 100,12,cubo.dtri);
	Teste(&creation);
	return 0;
}