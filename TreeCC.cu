#include "hip/hip_runtime.h"
#include "TreeCC.cuh"



    RGBA::RGBA() {
        this->r = 128;
        this->g = 128;
        this->b = 128;
        this->a = 255;
    };
    RGBA::RGBA(COR r, COR g, COR b, COR a) {
        this->r = r;
        this->g = g;
        this->b = b;
        this->a = a;
    };
    RGBA::RGBA(COR r, COR g, COR b) {
        this->r = r;
        this->g = g;
        this->b = b;
        this->a = 255;
    }
    RGBA& RGBA::operator=(const RGBA& other) {
        this->r = other.r;
        this->g = other.g;
        this->b = other.b;
        this->a = other.a;
        return *this;
    }
    RGBA& RGBA::operator+(const RGBA& other) {
        return RGBA(this->r + other.r,this->g + other.g,this->b + other.b,this->a + other.a);
    }
    RGBA& RGBA::operator*(const FLT m) {
        return RGBA( this->r * m,this->g * m,this->b * m ,this->a * m );
    }
    RGBA& RGBA::operator*=(const FLT to_mult) {
        this->r *= to_mult;
        this->g *= to_mult;
        this->b *= to_mult;
        this->a *= to_mult;
        return *this;
    }
    RGBAFLT::RGBAFLT() {

    }
    RGBAFLT::RGBAFLT(FLT v) {
        this->r = v;
        this->g = v;
        this->b = v;
        this->a = v;
    }
    RGBAFLT::RGBAFLT(FLT r,FLT g , FLT b , FLT a) {
        this->r = r;
        this->g = g;
        this->b = b;
        this->a = a;
    }

    RGBAFLT& RGBAFLT::operator=(const RGBAFLT& other) {
        this->r = other.r;
        this->g = other.g;
        this->b = other.b;
        this->a = other.a;
        return *this;
    }
    RGBAFLT& RGBAFLT::operator+(const RGBAFLT& other) {
        return RGBAFLT( this->r + other.r,this->g + other.g,this->b + other.b,this->a + other.a );
    }
    RGBAFLT& RGBAFLT::operator*(const FLT m) {
        return RGBAFLT(this->r * m,this->g * m,this->b * m ,this->a * m );
    }
    RGBAFLT& RGBAFLT::operator*(const RGBAFLT& m) {
        return RGBAFLT(this->r * m.r,this->g * m.g ,this->b* m.b ,this->a* m.a );
    }
    RGBAFLT& RGBAFLT::operator/(const FLT d) {
        if(d!=0)return RGBAFLT(this->r / d,this->g / d,this->b / d ,this->a / d );
        return *this;
    }
    RGBAFLT& RGBAFLT::operator*=(const FLT to_mult) {
        this->r *= to_mult;
        this->g *= to_mult;
        this->b *= to_mult;
        this->a *= to_mult;
        return *this;
    }
    RGBAFLT& RGBAFLT::operator/=(const FLT to_div) {
        this->r /= to_div;
        this->g /= to_div;
        this->b /= to_div;
        this->a /= to_div;
        return *this;
    }
    RGBAFLT& RGBAFLT::operator+=(const FLT to_add) {
        this->r += to_add;
        this->g += to_add;
        this->b += to_add;
        this->a += to_add;
        return *this;
    }
    RGBAFLT& RGBAFLT::operator+=(const RGBAFLT& to_add) {
        this->r += to_add.r;
        this->g += to_add.g;
        this->b += to_add.b;
        this->a += to_add.a;
        return *this;
    }


XYZ::XYZ() {

}
XYZ::XYZ(FLT x, FLT y, FLT z) {
    this->x = x;
    this->y = y;
    this->z = z;
}

XYZ& XYZ::operator +=(const XYZ& rhs) {
    this->x += rhs.x;
    this->y += rhs.y;
    this->z += rhs.z;
    return *this;
}
XYZ& XYZ::operator -=(const XYZ& rhs) {
    this->x -= rhs.x;
    this->y -= rhs.y;
    this->z -= rhs.z;
    return *this;
}



XYZ& XYZ::operator /=(const FLT& div) {
    if (div != 0) {
        this->x /= div;
        this->y /= div;
        this->z /= div;
    }
    return *this;
}

XYZ& XYZ::operator *=(const FLT& mult) {
    this->x *= mult;
    this->y *= mult;
    this->z *= mult;
    return *this;
}

XYZ& XYZ::operator =(const XYZ& rhs) {
    this->x = rhs.x;
    this->y = rhs.y;
    this->z = rhs.z;
    return *this;

}



XYZ& XYZ::operator +(const XYZ& rhs) {

    return XYZ(this->x + rhs.x,this->y + rhs.y,this->z + rhs.z);
}
XYZ& XYZ::operator -(const XYZ& rhs) {

    return XYZ(this->x - rhs.x,this->y - rhs.y,this->z - rhs.z );
}
XYZ& XYZ::operator /(const FLT& div) {
    if (div != 0)return XYZ(this->x / div,this->y / div,this->z / div );
    return XYZ(0,0,0 );
}

Angle::Angle(Spinner* spinner, short ptr) {
    this->r = 0.0;
    this->c = 1.0;
    this->s = 0.0;
    hipMemcpy(&this->c_ptr, &spinner->c[ptr % 3], sizeof(TRG), hipMemcpyDeviceToHost);
    hipMemcpy(&this->s_ptr, &spinner->s[ptr % 3], sizeof(TRG), hipMemcpyDeviceToHost);
    this->next = nullptr;

}
void Angle::set_next(Angle* next) {
    this->next = next;
    return;
}
void Angle::set_r(TRG r, UPXL p ) {
    this->r = r;
    this->c = COS(r);
    this->s = SIN(r);
    return;
}
void Angle::delta_r(TRG dr, UPXL p ) {
    this->r += dr;
    this->c = COS(this->r);
    this->s = SIN(this->r);
    return;
}
void Angle::update_spins() {
    hipMemcpy(this->c_ptr, &this->c, sizeof(TRG), hipMemcpyHostToDevice);
    hipMemcpy(this->s_ptr, &this->s, sizeof(TRG), hipMemcpyHostToDevice);
}
Plano& Plano::operator=(const Plano& other) {
    this->a = other.a;
    this->b = other.b;
    this->c = other.c;
    this->d = other.d;
    this->n = other.n;
    this->l = other.l;
    return *this;
}

Plano& Plano::operator*=(const FLT& mult) {
    this->a *= mult;
    this->b *= mult;
    this->c *= mult;
    this->d *= mult;
    this->n *= mult;
    return *this;
}

Particula::Particula() {

}

Particula& Particula::operator=(const Particula& p) {
    this->d = p.d;
    this->n_sub_prenchidas = p.n_sub_prenchidas;
    this->pos = p.pos;
    this->sub_particula = p.sub_particula;
    this->material = p.material;
    this->tri_ptr = p.tri_ptr;
    this->cor = p.cor;
    //this->luminosidade = p.luminosidade;
    return *this;
}

Pixel::Pixel() {

}
World::World() {

}

Creation::Creation(UPXL n_p,UPXL w, UPXL h,UPXL n_t,Triangulo* tri){
    this->area = w * h;
    this->h = h;
    this->w = w;
    this->n_pixel_render = n_p;
    //hipMalloc((void**)& this->world, sizeof(World));
    Frame frame;
    frame.w = w;
    frame.h = h;
    frame.a = w * h;
    hipMalloc((void**)&frame.buffer,4*w*h* sizeof(COR));
    Cam c;
    hipMalloc((void**)&c.frame, sizeof(Frame));
    hipMemcpy(c.frame, &frame, sizeof(Frame), hipMemcpyHostToDevice);
    //hipMemcpy(&c.frame->w, &frame.w, sizeof(UPXL), hipMemcpyHostToDevice);
    
    c.focus = 2.0;
    c.res = 50;
    c.dis = 100;
    //c.frame = df;
    //Cam* dc;
    World world;
    hipMalloc((void**)&world.cam, sizeof(Cam));
    hipMemcpy(world.cam, &c, sizeof(Cam), hipMemcpyHostToDevice);
    Light luz;
    luz.cor = RGBAFLT(1.0, 0.5, 0.0, 1.0);
    luz.energia = 3.0;
    luz.lenght = 1.0;
    XYZ lp = XYZ(3, 2, 1);

    hipMalloc((void**) & luz.pos, sizeof(XYZ));
    hipMemcpy(luz.pos, &lp, sizeof(XYZ), hipMemcpyHostToDevice);
    
    world.background = RGBAFLT(1.0, 1.0, 0.5, 0.4);
    world.sombra = RGBAFLT(0.0,0.0,0.0,1);
    world.n_cams = 1;
    world.n_lights = 1;
    world.n_tri = n_t;
    world.t_triangulos = tri;
    //world.cam = dc;
    world.n_reflections = 0;
    world.n_pixels = n_p;
    hipMalloc((void**)&world.lights, sizeof(Light));
    hipMemcpy(world.lights, &luz, sizeof(Light), hipMemcpyHostToDevice);
    //world.lights = nullptr;

    Pixel px; //= new Pixel[n_p];
   // for (UPXL i = 0; i < n_p; i += 1) {
        Particula* p = new Particula[n_t];
        for (UPXL j = 0; j < n_t; j += 1) {
            p[j].sub_particula = nullptr;
            p[j].tri_ptr = &tri[j];
            hipMalloc((void**)&p[j].shadows, n_t * sizeof(Material*));
        }
        hipMalloc((void**) &px.particulas, n_t * sizeof(Particula));
        hipMemcpy(px.particulas, p, n_t * sizeof(Particula), hipMemcpyHostToDevice);
    //}
        delete[]p;
    hipMalloc((void**)&world.pixels, sizeof(Pixel));
    hipMemcpy(world.pixels, &px, sizeof(Pixel), hipMemcpyHostToDevice);
    hipMalloc((void**) &this->world, sizeof(World));
    hipMemcpy(this->world, &world, sizeof(World), hipMemcpyHostToDevice);

}
namespace uteis {
    void set_spinner_next(Spinner* origem, Spinner* next) {
        hipMemcpy(&origem->next, next, sizeof(Spinner*), hipMemcpyDeviceToDevice);
    }
};

namespace devc {
    __device__ size_t Get_id() {
        size_t block_id = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
        size_t block_offset = block_id * blockDim.x * blockDim.y * blockDim.z;
        return block_offset + threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    }
    __device__ size_t Get_N_Thereads(size_t count) {
        return ((count < N_THREADS) && (count != 0)) ? count : N_THREADS;
    }
    __device__ FLT vetor_lenght(XYZ* v) {
        return SQRT(POW(v->x, 2) + POW(v->y, 2) + POW(v->z, 2));
    }
    __device__ FLT Vetor_lenght(XYZ* o, XYZ* f) {
        return SQRT(POW(f->x - o->x, 2) + POW(f->y - o->y, 2) + POW(f->z - o->z, 2));
    }
    __device__ bool entre_0_1(TRG n) {
        return ((n >= 0) && (n <= 1.0));
    }
    __device__ Material* get_material(Triangulo* tri, TRG a, TRG b, TRG c) {
        UPXL i = (tri->tex->height - 1) * (tri->w_h.h[0] * a + tri->w_h.h[1] * b + tri->w_h.h[2] * c) * tri->tex->width + (tri->tex->width - 1) * ((tri->w_h.w[0] * a + tri->w_h.w[1] * b + tri->w_h.w[2] * c));
        return &tri->tex->materials[i];
    }
    __global__ void calcular_vetor(Triangulo* tri_array, size_t count) {
        size_t id = Get_id();
        if (id < count) {
            Triangulo* tri = &tri_array[id];
            tri->vetor[0] = (*tri->ponto[1] - *tri->ponto[0]);
            tri->vetor[1] = (*tri->ponto[2] - *tri->ponto[1]);
            tri->vetor[2] = (*tri->ponto[0] - *tri->ponto[2]);
        }
        return;
    }
};

namespace first {
    FLT vetor_lenght(XYZ* v) {
        return SQRT(POW(v->x, 2) + POW(v->y, 2) + POW(v->z, 2));
    }
    void equacao_plano_host(Triangulo* tri) {

            tri->plano.a = (tri->ponto[1]->y - tri->ponto[0]->y) * (tri->ponto[2]->z - tri->ponto[0]->z) - (tri->ponto[2]->y - tri->ponto[0]->y) * (tri->ponto[1]->z - tri->ponto[0]->z);
            tri->plano.b = (tri->ponto[1]->z - tri->ponto[0]->z) * (tri->ponto[2]->x - tri->ponto[0]->x) - (tri->ponto[2]->z - tri->ponto[0]->z) * (tri->ponto[1]->x - tri->ponto[0]->x);
            tri->plano.c = (tri->ponto[1]->x - tri->ponto[0]->x) * (tri->ponto[2]->y - tri->ponto[0]->y) - (tri->ponto[2]->x - tri->ponto[0]->x) * (tri->ponto[1]->y - tri->ponto[0]->y);
            tri->plano.d = -(tri->plano.a * tri->ponto[0]->x + tri->plano.b * tri->ponto[0]->y + tri->plano.c * tri->ponto[0]->z);
            tri->plano.n = XYZ(tri->plano.a, tri->plano.b, tri->plano.c);
            tri->plano.l = first::vetor_lenght(&tri->plano.n);
            tri->plano.n /= tri->plano.l;
            printf("%f\n", tri->plano.b);
            printf("%f\n", tri->plano.l);
        
        return;
    }

    __global__ void equacao_plano(Triangulo* tri_array, size_t count) {
        size_t id = devc::Get_id();
        if (id < count) {
            Triangulo* tri = &tri_array[id];
            tri->plano.a = (tri->ponto[1]->y - tri->ponto[0]->y) * (tri->ponto[2]->z - tri->ponto[0]->z) - (tri->ponto[2]->y - tri->ponto[0]->y) * (tri->ponto[1]->z - tri->ponto[0]->z);
            tri->plano.b = (tri->ponto[1]->z - tri->ponto[0]->z) * (tri->ponto[2]->x - tri->ponto[0]->x) - (tri->ponto[2]->z - tri->ponto[0]->z) * (tri->ponto[1]->x - tri->ponto[0]->x);
            tri->plano.c = (tri->ponto[1]->x - tri->ponto[0]->x) * (tri->ponto[2]->y - tri->ponto[0]->y) - (tri->ponto[2]->x - tri->ponto[0]->x) * (tri->ponto[1]->y - tri->ponto[0]->y);
            tri->plano.d = -(tri->plano.a * tri->ponto[0]->x + tri->plano.b * tri->ponto[0]->y + tri->plano.c * tri->ponto[0]->z);
            tri->plano.n = XYZ(tri->plano.a, tri->plano.b, tri->plano.c);
            tri->plano.l = devc::vetor_lenght(&tri->plano.n);
            tri->plano.n /= tri->plano.l;
        }
        return;
    }

};

namespace spin {
    __device__ XYZ Spin(XYZ* src, Spinner* spinner) {
        return XYZ(src->x * spinner->c[0] * spinner->c[1] + src->y * spinner->s[1] + src->z * spinner->s[0], src->y * spinner->c[1] * spinner->c[2] - src->x * spinner->s[1] + src->z * spinner->s[2], src->z * spinner->c[0] * spinner->c[2] - src->x * spinner->s[1] - src->y * spinner->s[2] );
    }
    __global__ void plano_normal_lenght(Triangulo* tri_array, size_t count) {
        size_t id = devc::Get_id();
        if (id < count) {
            Triangulo* tri = &tri_array[id];
            tri->plano.l = SQRT(POW(tri->plano.a, 2) + POW(tri->plano.b, 2) + POW(tri->plano.c, 2));
        }
    }

    __global__ void equacao_plano(Triangulo* tri_array, size_t count) {
        size_t id = devc::Get_id();
        if (id < count) {
            Triangulo* tri = &tri_array[id];
            tri->plano.a = (tri->ponto[1]->y - tri->ponto[0]->y) * (tri->ponto[2]->z - tri->ponto[0]->z) - (tri->ponto[2]->y - tri->ponto[0]->y) * (tri->ponto[1]->z - tri->ponto[0]->z);
            tri->plano.b = (tri->ponto[1]->z - tri->ponto[0]->z) * (tri->ponto[2]->x - tri->ponto[0]->x) - (tri->ponto[2]->z - tri->ponto[0]->z) * (tri->ponto[1]->x - tri->ponto[0]->x);
            tri->plano.c = (tri->ponto[1]->x - tri->ponto[0]->x) * (tri->ponto[2]->y - tri->ponto[0]->y) - (tri->ponto[2]->x - tri->ponto[0]->x) * (tri->ponto[1]->y - tri->ponto[0]->y);
            tri->plano.d = -(tri->plano.a * tri->ponto[0]->x + tri->plano.b * tri->ponto[0]->y + tri->plano.c * tri->ponto[0]->z);
            tri->plano.n = (XYZ(tri->plano.a, tri->plano.b, tri->plano.c) / tri->plano.l);
        }
        return;
    }
    __global__ void Spin_vertexs_in_bone(Bone* bone) {
        size_t id = devc::Get_id();
        if (id < bone->n_vertex) {
            bone->vertexs[id].f = bone->vertexs[id].o;
            XYZ* src = &bone->vertexs[id].f;
            Spinner* cur_ang = nullptr;
            while (bone->father != nullptr) {
                bone = bone->father;
                cur_ang = bone->ang;
                while (cur_ang != nullptr) {
                    *src = Spin(src, cur_ang);
                    cur_ang = cur_ang->next;
                }
            }
            //*src = Spin(src, cam->angle);
        }
        return;

    }
    __global__ void Spin_bone(Bone* bone, size_t count) {
        size_t id = devc::Get_id();
        if (id < count) {
            Bone* cur_bone = &bone[id];
            bone[id].pos.f = bone[id].pos.o;
            XYZ* src = &bone[id].pos.f;
            Spinner* cur_ang;
            while (cur_bone->father != nullptr) {
                cur_bone = cur_bone->father;
                cur_ang = bone->ang;
                while (cur_ang != nullptr) {
                    *src = Spin(src, cur_ang);
                    cur_ang = cur_ang->next;
                }

            }
            
            Spin_vertexs_in_bone <<<BLOCKS(bone[id].n_vertex), N_THREADS >> > (&bone[id]);
            
            //hipDeviceSynchronize();
            
            //hipDeviceSynchronize();
            
        }
        return;
    }
    __global__ void Spin_vertexs_in_bone_cam(Bone* bone, Bone* cam) {
        size_t id = devc::Get_id();
        if (id < bone->n_vertex) {
            bone->vertexs[id].c = bone->vertexs[id].f;
            Spinner* cur_ang = cam->ang;
            while (cur_ang != nullptr) {
                bone->vertexs[id].c = Spin(&bone->vertexs[id].c, cur_ang);
                cur_ang = cur_ang->next;
            }
        }
        return;
    }
    __global__ void Spin_Bone_Cam(Bone* bone, Bone* cam, size_t count) {
        size_t id = devc::Get_id();
        if (id < count) {
            bone[id].pos.c = bone[id].pos.f;
            Spinner* cur_ang = cam->ang;
            while (cur_ang != nullptr) {
                bone[id].pos.c = Spin(&bone[id].pos.c, cur_ang);
                cur_ang = cur_ang->next;
            }
        }
        Spin_vertexs_in_bone_cam << <BLOCKS(bone[id].n_vertex), N_THREADS >> > (&bone[id], cam);
        //hipDeviceSynchronize();

    }
    __global__ void vertex_add(Bone* bone, FLT focus) {
        //size_t count = bone->n_points;
        size_t id = devc::Get_id();
        if (id < bone->n_vertex)bone->vertexs[id].c = XYZ(bone->vertexs[id].c.x * PRJ(bone->origem.y + bone->vertexs[id].c.y,focus) + bone->origem.x,bone->origem.y + bone->vertexs[id].c.y ,bone->vertexs[id].c.z * PRJ(bone->origem.y + bone->vertexs[id].c.y,focus) + bone->origem.z );

        return;
    }
    __global__ void bone_add(Bone* bone, Cam* cam, size_t count) {
        size_t id = devc::Get_id();
        if (id < count) {
            Bone* cur_bone = &bone[id];
            XYZ* src = &bone[id].origem;
            *src = bone[id].pos.c;
            while (bone->father != nullptr) {
                cur_bone = cur_bone->father;
                *src += cur_bone->pos.c;
            }
            *src -= cam->center->pos.c;
            *src = XYZ(src->x * PRJ(src->y,cam->focus),src->y,src->z * PRJ(src->y,cam->focus) );
            vertex_add << <BLOCKS(bone[id].n_vertex), N_THREADS >> > (&bone[id], cam->focus);
            //hipDeviceSynchronize();
            //ponto_add<<<BLOCKS(bone[id].n_points), N_THREADS >> > (&bone[id], cam->focus);
            //ponto_add <<<BLOCKS(bone[id]->n_points), N_THREADS >>> (&bone[id], focus);
        }
        return;

    }
};

namespace ray {
    
    __device__ TRG Calculate_baricentro(XYZ* I, XYZ* O, XYZ* P) {
       
        FLT dot_product = (O->x * P->x + O->y * P->y + O->z * P->z) / (P->x * P->x + P->y * P->y + P->z * P->z);//(POW(P->x, 2) + POW(P->y, 2) + POW(P->z, 2));
        
        XYZ v = XYZ(O->x - dot_product * P->x, O->y - dot_product * P->y, O->z - dot_product * P->z);
        
        return 1.0 - ((v.x * I->x + v.y * I->y + v.z * I->z) / (v.x * O->x + v.y * O->y + v.z * O->z));
    }
    __device__ bool sameside(TRG* p,XYZ C,Triangulo* tri,Plano* plano,short a,short b){
        XYZ AB = *tri->ponto[b] - *tri->ponto[a];
        XYZ AC = C -*tri->ponto[a];
        
        FLT crossProductX = AB.y * AC.z - AB.z * AC.y; // Componente Z da normal
        FLT crossProductY = AB.z * AC.x - AB.x * AC.z;
        FLT crossProductZ = AB.x * AC.y - AB.y * AC.x;
        *p = SQRT(crossProductX * crossProductX + crossProductY * crossProductY + crossProductZ * crossProductZ);
        //printf("Foi");
        return (crossProductZ * plano->c >= 0) && (crossProductY * plano->b >= 0) && (crossProductX * plano->a >= 0);

    }
    __device__ bool Metodo_separacao(XYZ vt,Triangulo* tri,Plano* plano,TRG*a,TRG*b,TRG*c) {
        return sameside(a, vt, tri, plano, 0, 1) && sameside(a, vt, tri, plano, 1, 2) && sameside(a, vt, tri, plano, 2, 0);
    }
    __device__ RGBAFLT cor_sub_particulas(Particula* particula,RGBAFLT* background) {
        
        RGBAFLT result = *background;
        
        for (UPXL i = 0; i < particula->n_sub_prenchidas; i += 1) {
        
            result.a = result.a + (1 - result.a) * particula->sub_particula[i].cor.a;
            result.r = result.r * (1 - particula->sub_particula[i].cor.a) + particula->sub_particula[i].cor.r* particula->sub_particula[i].cor.a;
            result.g = result.g * (1 - particula->sub_particula[i].cor.a) + particula->sub_particula[i].cor.g * particula->sub_particula[i].cor.a;
            result.b = result.b * (1 - particula->sub_particula[i].cor.a) + particula->sub_particula[i].cor.b * particula->sub_particula[i].cor.a;
        
        }
        return result;
    }
   
    void pixel_color(World* world, Pixel* pixel,UPXL n) {
        Pixel p;
        hipMemcpy(&p, pixel, sizeof(Pixel), hipMemcpyDeviceToHost);
        p.cor = world->background;
        //hipMemcpy(&p.cor, &world->background,sizeof(RGBAFLT), hipMemcpyDeviceToHost);
        
        for (UPXL i = 0; i < n; i += 1) {
            Particula aux;
            hipMemcpy(&aux, &p.particulas[i], sizeof(Particula), hipMemcpyDeviceToHost);
            printf("erro aqui %d\n",i);
            p.cor.r = p.cor.r * (1 - aux.cor.a) + aux.cor.r * aux.cor.a;
            p.cor.g = p.cor.g * (1 - aux.cor.a) + aux.cor.g * aux.cor.a;
            p.cor.b = p.cor.b * (1 - aux.cor.a) + aux.cor.b * aux.cor.a;
            p.cor.a = p.cor.a + (1 - p.cor.a) * aux.cor.a;
            
        }
        
        hipMemcpy(pixel, &p, sizeof(Pixel), hipMemcpyHostToDevice);
        return;
    }

    __device__ void Produto_lights(World* world,Particula* particula) {
        
        RGBAFLT luminosidade = world->sombra;
        
        RGBAFLT e_result;// = RGBAFLT(1.0);
        for (UPXL li = 0; li < world->n_lights; li += 1) {
            
            e_result = world->lights[li].cor;
            for (UPXL sp = 0; sp < world->n_tri && e_result.a >0; sp += 1) {
                
                if (particula->shadows[sp] != nullptr) {

                    e_result.a *= (1.0 - (particula[sp].material->cor.a * particula->shadows[sp]->ipermeability));
                    e_result.r *= e_result.a * particula->shadows[sp]->cor.r;
                    e_result.g *= e_result.a * particula->shadows[sp]->cor.g;
                    e_result.b *= e_result.a * particula->shadows[sp]->cor.b;
                }
            }

            luminosidade += e_result* PRJ(world->lights[li].energia, world->lights[li].lenght * devc::Vetor_lenght(world->lights[li].pos, &particula->pos));
            
            //e_result = { 1.0 };
        
        }
        particula->cor =  ((particula->cor)* (luminosidade / (world->n_lights + 1.0)))* (1.0 - particula->material->reflect);
        
        if (particula->material->reflect > 0 && particula->sub_particula != nullptr) {
            
            particula->cor += cor_sub_particulas(particula,&world->background)* particula->material->reflect;
        }
        return;
    }

    __global__ void shadow_intersection(World* world, Particula* particula, UPXL light_id) {

        size_t id = devc::Get_id();

        if (id < (world->n_tri)) {
            printf("%d\n",id);
            particula->shadows[id] = nullptr;
            //particula->shadows[light_id * world->n_tri + id] = nullptr;
            
            if (particula->tri_ptr != &world->t_triangulos[id]) {
                
                Triangulo* tri = &world->t_triangulos[id];

                FLT div = tri->plano.a * (world->lights[light_id].pos->x - particula->pos.x) + tri->plano.b * (world->lights[light_id].pos->y - particula->pos.y) + tri->plano.c * (world->lights[light_id].pos->z - particula->pos.z);
                
                if (div != 0) {

                    FLT t = -(tri->plano.a * particula->pos.x + tri->plano.b * particula->pos.y + tri->plano.c * particula->pos.z + tri->plano.d) / div;

                    if (devc::entre_0_1(t)) {
                        
                        XYZ P = XYZ(particula->pos.x + t * ((world->lights[light_id].pos->x - particula->pos.x)), particula->pos.y + t * (world->lights[light_id].pos->y - particula->pos.y), particula->pos.z + t * (world->lights[light_id].pos->z - particula->pos.z));
                        TRG a, b, c;
                        if (Metodo_separacao(P, tri, &tri->plano, &a, &b, &c)) {
                            if ((a+b+c)> 0) {
                                FLT s = (a + b + c);
                                a /= s;
                                b /= s;
                                c /= s;
                            }
                            printf("%d a:%f b:%f c:%f soma:%f\n", id, a, b, c, a + b + c);
                            particula->shadows[id] = devc::get_material(tri, a, b, c);
                            //particula->shadows[light_id * world->n_tri + id] = devc::get_material(tri, a, b, c);
                            
                        }
                        /* XYZ AP = P - *tri->ponto[0];
                        XYZ BP = P - *tri->ponto[1];
                        XYZ CP = P - *tri->ponto[2];

                        TRG a = Calculate_baricentro(&AP, &tri->vetor[0], &tri->vetor[1]);
                        TRG b = Calculate_baricentro(&BP, &tri->vetor[1], &tri->vetor[2]);
                        TRG c = Calculate_baricentro(&CP, &tri->vetor[2], &tri->vetor[0]);

                        if ((a >= 0 && b >= 0 && c >= 0))particula->shadows[light_id * world->n_tri + id] = devc::get_material(tri, a, b, c);  */

                    }
                }
            }
            return;
        }
    }

    __global__ void light_ray(World* world, Particula* particula,size_t count) {
        
        size_t id = devc::Get_id();
        
        if (id < count) {
            
            for (UPXL li = 0; li < world->n_lights; li += 1) {
                printf("%d light ray:", li);
                shadow_intersection << <BLOCKS(world->n_tri), N_THREADS >> > (world, &particula[id], li);
                SYNCTHREADS();
                
                //
            }
            printf("Aqui");
            
            
            Produto_lights(world, &particula[id]);
        }
        return;
    }

    __global__ void ordenar_sub_particulas(World* world, Particula* particula) {
        
        size_t id = devc::Get_id();
        
        if (id < world->n_tri) {
        
            particula->n_sub_prenchidas = world->n_tri;
            
            UPXL n_particulas_vazias = 0;
            FLT a = 0.0;
            Particula aux;
            
            for (UPXL i = 0; i < (world->n_tri - (n_particulas_vazias + 1)); i += 1) {
                
                for (UPXL j = i + 1; j < (world->n_tri - n_particulas_vazias); j += 1) {
                
                    if (particula->sub_particula[i].material == nullptr) {
                    
                        aux = particula->sub_particula[i];
                        particula->sub_particula[i] = particula->sub_particula[world->n_tri - j];
                        particula->sub_particula[world->n_tri - j] = aux;
                        
                        n_particulas_vazias += 1;
                    
                    }
                    
                    if ((particula->sub_particula[i].material == nullptr) || particula->sub_particula[j].d < particula->sub_particula[i].d) {
                       
                        aux = particula->sub_particula[i];
                        particula->sub_particula[i] = particula->sub_particula[j];
                        particula->sub_particula[j] = aux;

                    }

                }
                if (particula->sub_particula[i].material == nullptr) {

                    particula->n_sub_prenchidas -= i;
                    i = world->n_tri;

                }
                else {

                    a = (1 - a ) * particula->sub_particula[i].material->cor.a + a;

                    if (a >=1.0) {

                        particula->n_sub_prenchidas -= (i + 1);
                        i = world->n_tri;
                    }
                }
            }
            particula->n_sub_prenchidas = world->n_tri - (particula->n_sub_prenchidas);

        }
        return;
    }

    __global__ void traycing_sub_particulas(World* world, Particula* particula) {
        
        size_t id = devc::Get_id();
        
        if (id < world->n_tri) {
            
            particula->sub_particula[id].material = nullptr;
            
            if (particula->sub_particula[id].tri_ptr == particula->tri_ptr)return;
            
            Triangulo* s_tri = particula->sub_particula[id].tri_ptr;
            Plano p_plano = particula->tri_ptr->plano;
            Plano s_plano = particula->sub_particula[id].tri_ptr->plano;
            
            if (particula->fliped)p_plano *= -1;
            
            particula->sub_particula[id].fliped = (signbit(s_plano.b) == signbit(p_plano.b));
            
            if (particula->sub_particula[id].fliped)s_plano *= -1.0;

            FLT div = (p_plano.n.x * world->cam->dis) * s_plano.a + (p_plano.n.y * world->cam->dis) * s_plano.b + (p_plano.n.z * world->cam->dis) * s_plano.c;
            
            if (div != 0) {
                
                TRG t = -(s_plano.a * particula->pos.x + s_plano.b * particula->pos.y + s_plano.c * particula->pos.z + s_plano.d) / div;
                
                if (devc::entre_0_1(t)) {
                    
                    XYZ P = XYZ(particula->pos.x + t * (p_plano.n.x * world->cam->dis),particula->pos.y + t * (p_plano.n.y * world->cam->dis),particula->pos.z + t * (p_plano.n.z * world->cam->dis) );
                    XYZ AP = P - *s_tri->ponto[0];
                    XYZ BP = P - *s_tri->ponto[1];
                    XYZ CP = P - *s_tri->ponto[2];
                    
                    TRG a = Calculate_baricentro(&AP, &s_tri->vetor[0], &s_tri->vetor[1]);
                    TRG b = Calculate_baricentro(&BP, &s_tri->vetor[1], &s_tri->vetor[2]);
                    TRG c = Calculate_baricentro(&CP, &s_tri->vetor[2], &s_tri->vetor[0]);
                    
                    if ((a >= 0 && b >= 0 && c >= 0)) {
                        particula->sub_particula[id].pos = P;
                        particula->sub_particula[id].d = world->cam->dis * t;
                        particula->sub_particula[id].material = devc::get_material(s_tri, a, b, c);
                        particula->sub_particula[id].cor = particula->sub_particula[id].material->cor;
                    }
                }
            }
        }
    }
    
    __global__ void traycing_sub_em_sub(World* world,Particula* particula,UPXL nivel){
        
        size_t id = devc::Get_id();
        
        if (id < particula->n_sub_prenchidas) {
            
            if (particula->sub_particula[id].material->reflect > 0) {
        
                traycing_sub_particulas<<<BLOCKS(world->n_tri),N_THREADS >> >(world, &particula->sub_particula[id]);//<<<n_tri-1>>>
                //hipDeviceSynchronize();
                SYNCTHREADS();
                ordenar_sub_particulas << <BLOCKS(world->n_tri), N_THREADS >> > (world, &particula->sub_particula[id]);//<<<n_tri-1>>>
                //hipDeviceSynchronize();
                SYNCTHREADS();
                if ((nivel > 1) && (particula->sub_particula[id].n_sub_prenchidas > 0)) {
                
                    traycing_sub_em_sub<<<BLOCKS(particula->sub_particula[id].n_sub_prenchidas),N_THREADS>>>(world, &particula->sub_particula[id],nivel-1);//<<<n_tri-1>>>
                    //hipDeviceSynchronize();
                    SYNCTHREADS();
                }
                
                light_ray<<<BLOCKS(particula->sub_particula[id].n_sub_prenchidas),N_THREADS>>>(world, &particula->sub_particula[id], particula->n_sub_prenchidas);
                SYNCTHREADS();
                //hipDeviceSynchronize();
                
            }
        }

    }
    
    __global__ void traycing_sub_particulas_em_pixels(World* world, Pixel* pixel) {
        
        size_t id = devc::Get_id();
        
        if (id < pixel->n_particulas_prenchidas) {
        
            if ((pixel->particulas[id].material->reflect > 0) && (pixel->particulas[id].sub_particula != nullptr)) {
            
                traycing_sub_particulas<<<BLOCKS(world->n_tri),N_THREADS>> >(world, &pixel->particulas[id]);//<<<n_tri-1>>>
                SYNCTHREADS();
                //hipDeviceSynchronize();
                
                ordenar_sub_particulas << <BLOCKS(world->n_tri ), N_THREADS >> > (world, &pixel->particulas[id]);//<<<n_tri-1>>>
                SYNCTHREADS();
                //hipDeviceSynchronize();
                
                if ((world->n_reflections > 1 )&& (pixel->particulas[id].n_sub_prenchidas >0)) {
                    
                    traycing_sub_em_sub << <BLOCKS(pixel->n_particulas_prenchidas), N_THREADS >> > (world, &pixel->particulas[id], world->n_reflections - 1);//<<<n_tri-1>>>
                    SYNCTHREADS();
                
                }
                
                light_ray <<<BLOCKS(pixel->n_particulas_prenchidas), N_THREADS >> > (world, pixel->particulas[id].sub_particula, pixel->particulas[id].n_sub_prenchidas);
                SYNCTHREADS();
                
            }
        }
        return;
    }

    /*__global__ void ordenar_particulas_em_pixels(World* world, Pixel* pixel) {
        
        size_t id = devc::Get_id();
        
        if (id < world->n_tri) {
            
            pixel->n_particulas_prenchidas = world->n_tri;
            
            FLT a = 0;
            UPXL n_particulas_vazias = 0;
            Particula aux;           
            
            for (UPXL i = 0; i < (world->n_tri - (n_particulas_vazias + 1)); i += 1) {
                
                for (UPXL j = i + 1; j < (world->n_tri - n_particulas_vazias); j += 1) {
                    
                    if (pixel->particulas[i].material == nullptr) {
                        
                        aux = pixel->particulas[i];
                        pixel->particulas[i] = pixel->particulas[world->n_tri - j];
                        pixel->particulas[world->n_tri - j] = aux;
                        n_particulas_vazias += 1;

                    }
                    if ((pixel->particulas[i].material == nullptr) || (pixel->particulas[j].d < pixel->particulas[i].d)) {
                        
                        aux = pixel->particulas[i];
                        pixel->particulas[i] = pixel->particulas[j];
                        pixel->particulas[j] = aux;

                    }

                }
                if (pixel->particulas[i].material == nullptr) {

                    pixel->n_particulas_prenchidas -= i;
                    i = world->n_tri;

                }
                else {

                    a = (1 - a) * pixel->particulas[i].material->cor.a + a;

                    if (a >= 255) {

                        pixel->n_particulas_prenchidas -= (i + 1);
                        i = world->n_tri;

                    }
                }
            }
            pixel->n_particulas_prenchidas = world->n_tri - (pixel->n_particulas_prenchidas);
        }
        return;
    }*/
    void ordenar_particulas_em_pixels(World* w, Pixel* p) {
        
            p->n_particulas_prenchidas = w->n_tri;

            FLT a = 0;

            //UPXL n_particulas_vazias = 0;
            Particula pi;
            Particula pj;
            Particula pnj;
            Particula aux;

            for (UPXL i = 0; i <(p->n_particulas_prenchidas - 1); i += 1) {
               // printf("i:%d\n", i);
                hipMemcpy(&pi, &p->particulas[i], sizeof(Particula), hipMemcpyDeviceToHost);
                for (UPXL j = i + 1; j < (p->n_particulas_prenchidas); j += 1) {
                   // printf("j:%d\n", j);
                    hipMemcpy(&pj, &p->particulas[j], sizeof(Particula), hipMemcpyDeviceToHost);
                    if (pi.material == nullptr) {

                        aux = pi;
                        hipMemcpy(&p->particulas[i], &p->particulas[w->n_tri - j], sizeof(Particula), hipMemcpyDeviceToDevice);
                        hipMemcpy(&p->particulas[w->n_tri - j],&aux, sizeof(Particula), hipMemcpyHostToDevice);
                        p->n_particulas_prenchidas -= 1;

                    }
                    //if (pi.material != nullptr || pj.material != nullptr)printf("Aqui");
                    if ((pi.material == nullptr) || ((pj.material!= nullptr)&&(pj.d < pi.d))) {
                        
                        aux = pi;
                        hipMemcpy(&p->particulas[i], &p->particulas[j], sizeof(Particula), hipMemcpyDeviceToDevice);
                        hipMemcpy(&p->particulas[j], &aux, sizeof(Particula), hipMemcpyHostToDevice);

                    }
                    hipMemcpy(&pi, &p->particulas[i], sizeof(Particula), hipMemcpyDeviceToHost);
                }
                //hipMemcpy(&pi, &p->particulas[i], sizeof(Particula), hipMemcpyDeviceToHost);
                
                if (pi.material == nullptr) {
                    
                    p->n_particulas_prenchidas = i;
                    i = w->n_tri;

                }
                else {
                    //printf("erro aqui");
                    Material m;
                    hipMemcpy(&m, pi.material, sizeof(Material), hipMemcpyDeviceToHost);
                    a = (1 - a) * m.cor.a + a;

                    if (a >= 1.0) {

                        p->n_particulas_prenchidas = (i+1);
                        i = w->n_tri;
                        //printf("erro aqui2 ");

                    }
                }
            }
            
            if (p->n_particulas_prenchidas == 0)printf("\nBolacha\n");
        return;
    }

    __global__ void traycing_particulas_em_pixels(World* world, Pixel* pixel) {

        size_t id = devc::Get_id();
        
        if (id < world->n_tri) {
            
            Triangulo* tri = pixel->particulas[id].tri_ptr;//&world->t_triangulos[id];
            //printf("%p\n", tri);
            pixel->particulas[id].material = nullptr;
            pixel->particulas[id].d = world->cam->dis;
            Plano plano;
            plano = tri->plano;

            pixel->particulas[id].fliped = !signbit(plano.b);

            if (pixel->particulas[id].fliped) {
                plano *= -1.0;//flip caso a normal esteja apontando para a mesma dire��o que o ray
                //if (!(id % 100))printf("%f\n", plano.b);
            }
            FLT div = world->cam->dis * plano.b;
            //if(!(id%100))printf("%f\n", plano.b);
            if (div != 0) {
                //printf("%d", id);
                TRG t = -(plano.a * pixel->w + plano.c * pixel->h + plano.d) / div;
               
                if (devc::entre_0_1(t)) {

                    //printf("%d\n", id);
                    
                    XYZ P = XYZ(pixel->w, t * world->cam->dis , pixel->h);
                   
                    //XYZ AP = XYZ( pixel->w - tri->ponto[0]->x,t * world->cam->dis - tri->ponto[0]->y, pixel->h - tri->ponto[0]->z );
                    //XYZ BP = XYZ(pixel->w - tri->ponto[1]->x,t * world->cam->dis - tri->ponto[1]->y, pixel->h - tri->ponto[1]->z );
                    //XYZ CP = XYZ(pixel->w - tri->ponto[2]->x,t * world->cam->dis - tri->ponto[2]->y, pixel->h - tri->ponto[2]->z );
                    
                    //XYZ AB = *tri->ponto[1] - *tri->ponto[0];
                    //XYZ BC = *tri->ponto[2] - *tri->ponto[1];
                    //XYZ CA = *tri->ponto[0] - *tri->ponto[2];
                    //TRG a = Calculate_baricentro(&AP, &AB, &BC);
                    //TRG b = Calculate_baricentro(&BP, &CA, &AB);
                    //TRG c = Calculate_baricentro(&CP, &tri->vetor[2], &tri->vetor[0]);
                    //tri->vetor[0] = tri->ponto[0] - tri->ponto[1];
                    //TRG a = Calculate_baricentro(&AP, &tri->vetor[0], &tri->vetor[1]);
                    //TRG b = Calculate_baricentro(&BP, &tri->vetor[2], &tri->vetor[0]);
                    //TRG c = Calculate_baricentro(&CP, &tri->vetor[2], &tri->vetor[0]);
                    //printf("a: %f, b: %f, c: %f\n", a, b, c);
                    TRG a, b, c;
                    if(Metodo_separacao(P,tri,&plano,&a,&b,&c)){
                    //if ((a >= 0) && (b >= 0) && (a<=1) && (b<=1)){
                        //printf("aqui");
                        //TRG c = 1 - (a + b);
                        if (plano.l > 0) {
                            a /= plano.l;
                            b /= plano.l;
                            c /= plano.l;
                        }
                        //printf("\n%f\n", (a + b + c));
                        //if ((a + b + c) == 0.5) {
                            pixel->particulas[id].pos = XYZ(pixel->w, world->cam->dis * t, pixel->h);
                            pixel->particulas[id].d = world->cam->dis * t;
                            pixel->particulas[id].material = devc::get_material(tri, a, b, c);
                            pixel->particulas[id].cor = pixel->particulas[id].material->cor;
                        //}
                        //printf("cor %f", pixel->particulas[id].cor.b);

                    }
                }
            }
        }
        return;
    }

    void traycing_pixels(Creation* c) {

        World world;
        
        Cam cam;
        Frame frame;
        Pixel p;
        hipMemcpy(&world, c->world, sizeof(World), hipMemcpyDeviceToHost);
        hipMemcpy(&cam, world.cam, sizeof(Cam), hipMemcpyDeviceToHost);
        hipMemcpy(&frame, cam.frame, sizeof(Frame), hipMemcpyDeviceToHost);
        hipMemcpy(&p, world.pixels, sizeof(Pixel), hipMemcpyDeviceToHost);
        COR* buffer = new COR[4*frame.a];
        //for(UPXL y = 0;y< frame.h;y+=1)for (UPXL id = 0; id < frame.w;id+=1) {
            //std::cout << id << std::endl;
        for (UPXL id = 0; id < frame.a; id += 1) {
            //printf("\nid: %d\n",id);
            //p.h = ((id / frame.w) - ((frame.h/2.0)))/(cam.res);
            //p.w = ((id % frame.w) - ((frame.w/2.0)))/(cam.res);
            p.w = (id % frame.w);
            p.h = (id / frame.w);
            p.w /= cam.res;
            p.h /= cam.res;
            p.w -= frame.w / (2.0*cam.res);
            p.h -= frame.h / (2.0 * cam.res);
            
               //p.h = y/cam.res ;
               // p.w = id/cam.res ;
            
            hipMemcpy(world.pixels, &p, sizeof(Pixel), hipMemcpyHostToDevice);
            //hipMemcpy(&p.h,&h, sizeof(UPXL), hipMemcpyHostToDevice);
            //hipMemcpy(&p.w, &w, sizeof(UPXL), hipMemcpyHostToDevice);
            traycing_particulas_em_pixels << <BLOCKS(world.n_tri), N_THREADS >> > (c->world, world.pixels);
            //
            hipDeviceSynchronize();
            hipMemcpy(&p, world.pixels, sizeof(Pixel), hipMemcpyDeviceToHost);
            hipMemcpy(&world, c->world, sizeof(World), hipMemcpyDeviceToHost);
            
            ordenar_particulas_em_pixels(&world, &p);
            //SYNCTHREADS();
            hipDeviceSynchronize();
            //hipMemcpy(&world, c->world, sizeof(World), hipMemcpyDeviceToHost);
            //hipMemcpy(&p, world.pixels, sizeof(Pixel), hipMemcpyDeviceToHost);
            //hipMemcpy(&world, c->world, sizeof(World), hipMemcpyDeviceToHost);
            //hipMemcpy(&p, world.pixels, sizeof(Pixel), hipMemcpyDeviceToHost);
            hipMemcpy(world.pixels, &p, sizeof(Pixel), hipMemcpyHostToDevice);
            if (world.n_reflections > 0) {
                
                traycing_sub_particulas_em_pixels << <BLOCKS(p.n_particulas_prenchidas), N_THREADS >> > (c->world, world.pixels);
                //SYNCTHREADS();
                hipDeviceSynchronize();

            }
            
            if (p.n_particulas_prenchidas > 0) {
                //if (p.n_particulas_prenchidas > 1)printf("h:%f , w: %f\n",p.h,p.w);
                //hipMemcpy(&p, world.pixels, sizeof(Pixel), hipMemcpyDeviceToHost);
                printf("Erro 0  aqui");
                light_ray << <BLOCKS(p.n_particulas_prenchidas), N_THREADS >> > (c->world, p.particulas, p.n_particulas_prenchidas);

                //SYNCTHREADS();
                hipDeviceSynchronize();
                
                
            }
            hipMemcpy(world.pixels, &p, sizeof(Pixel), hipMemcpyHostToDevice);
            //hipMemcpy(world.pixels, &p, sizeof(Pixel), hipMemcpyHostToDevice);
            printf("Erro 1  aqui");
            pixel_color(&world, world.pixels,p.n_particulas_prenchidas);
            //seta cor em frame
            Pixel px;
            printf("Erro 2  aqui");
            hipMemcpy(&px, world.pixels, sizeof(Pixel), hipMemcpyDeviceToHost);
            //printf("Erro 1  aqui");
            buffer[4 * id] = (COR)ceil(255 * px.cor.b);
            buffer[1 + 4 * id] = (COR)ceil(255 * px.cor.g);
            buffer[2 + 4 * id ] = (COR)(255 * px.cor.r);
            buffer[3 + 4 * id ] = (COR)(255 * px.cor.a);
            ///if (buffer[2 + 4 * id] == 0)printf("%d", id);
           /* buffer[4 * (id + y * frame.w)] = (COR)ceil(255 * px.cor.b);
            buffer[1 + 4 * (id + y * frame.w)] = (COR)ceil(255 * px.cor.g );
            buffer[2 + 4 * (id + y * frame.w)] = (COR)(255 * px.cor.r );
            buffer[3 + 4 * (id + y * frame.w)] = (COR)(255 * px.cor.a);*/
            //std::cout << ceil(255 * px.cor.b ) << std::endl;

            //printf("Erro 1  aqui");
        }
        hipMemcpy(frame.buffer, buffer, 4 * frame.a* sizeof(COR), hipMemcpyHostToDevice);
        delete[] buffer;
        return;
    }

       /* __global__ void traycing_pixels(World* world,UPXL p) {

        size_t id = devc::Get_id();

        if ((id < world->n_pixels) && ((id+p* world->n_pixels) < world->cam->frame->a)) {

            world->pixels[id].h = (((id+p* world->n_pixels) / world->cam->frame->w) - (world->cam->frame->h / 2)) / world->cam->res;
            world->pixels[id].w = (((id+p* world->n_pixels) % world->cam->frame->w) - (world->cam->frame->w / 2)) / world->cam->res;

            traycing_particulas_em_pixels <<<BLOCKS(world->n_tri), N_THREADS >> > (world, &world->pixels[id]);
            SYNCTHREADS();
            //hipDeviceSynchronize();

            ordenar_particulas_em_pixels << <BLOCKS(world->n_tri), N_THREADS >> > (world, &world->pixels[id]);
            SYNCTHREADS();
            //hipDeviceSynchronize();

            if (world->n_reflections > 0) {
                traycing_sub_particulas_em_pixels<<<BLOCKS(world->pixels[id].n_particulas_prenchidas),N_THREADS>>>(world, &world->pixels[id]);
                SYNCTHREADS();
                //hipDeviceSynchronize();
            }

            light_ray << <BLOCKS(world->pixels[id].n_particulas_prenchidas), N_THREADS >> > (world, world->pixels[id].particulas, world->pixels->n_particulas_prenchidas);
            SYNCTHREADS();
            //hipDeviceSynchronize();

            pixel_color(world, &world->pixels[id]);
            //seta cor em frame
            world->cam->frame->buffer[    4*(id+ p * world->n_pixels)] = ceil(255 * world->pixels->cor.b);
            world->cam->frame->buffer[1 + 4*(id+ p * world->n_pixels)] = ceil(255 * world->pixels->cor.g);
            world->cam->frame->buffer[2 + 4*(id+ p * world->n_pixels)] = ceil(255 * world->pixels->cor.r);
            world->cam->frame->buffer[3 + 4*(id+ p * world->n_pixels)] = ceil(255 * world->pixels->cor.a);

        }
        return;
    }
    */
};
//mudar negocio do world ser em device
void Render(Creation* creation) {
    //for (UPXL p = 0; p < creation->area; p += 1) {
        ray::traycing_pixels(creation);
    //}
}

